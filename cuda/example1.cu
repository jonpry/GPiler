// example1.cu : Defines the entry point for the console application.
//
 
#include <stdio.h>
#include <hip/hip_runtime.h>

//Place holder function so we can redirect the call during link 
__device__  __noinline__ void mapitfoo(int N, int *out, int *in){
	out[N] = in[N] * in[N];
}

// Kernel that executes on the CUDA device
__global__ void square_array(int N, int *out, int *in)
{
  for(int i=0; i<N; i+=blockDim.x*gridDim.x){
  	int idx = i + blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<N)
		mapitfoo(idx,out,in);
  }
}
 
// main routine that executes on the host
int main(void)
{
  int *in_h, *in_d, *out_h, *out_d;  // Pointer to host & device arrays
  const int N = 10;  // Number of elements in arrays


  size_t size = N * sizeof(int);
  in_h = (int *)malloc(size);        // Allocate array on host
  out_h = (int *)malloc(size);        // Allocate array on host

  hipMalloc((void **) &in_d, size);   // Allocate array on device
  hipMalloc((void **) &out_d, size);   // Allocate array on device


  // Initialize host array and copy it to CUDA device
  for (int i=0; i<N; i++) in_h[i] = i;
  hipMemcpy(in_d, in_h, size, hipMemcpyHostToDevice);

  // Do calculation on device:
  int block_size = 32; //TODO: get this number from device info
  int n_blocks = 1; //TODO: determine number of SM's
  square_array <<< n_blocks, block_size >>> (N, out_d, in_d);
  // Retrieve result from device and store it in host array

  hipMemcpy(out_h, out_d, sizeof(int)*N, hipMemcpyDeviceToHost);
  
  // Print results
  for (int i=0; i<N; i++) printf("%d %d %d\n", i, in_h[i], out_h[i]);
  // Cleanup
  free(in_h); 
  free(out_h);
  hipFree(in_d);
  hipFree(out_d);
}
