/* 
GPiler - example1.cu
Copyright (C) 2013 Jon Pry and Charles Cooper

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/
 
#include <stdio.h>
#include <hip/hip_runtime.h>

//Place holder function so we can redirect the call during link 
__device__  __noinline__ void mapitfoo(int N, int *out, int *in){
	out[N] = in[N] * in[N];
}

// Kernel that executes on the CUDA device
__global__ void square_array(int N, int *out, int *in)
{
  for(int i=0; i<N; i+=blockDim.x*gridDim.x){
  	int idx = i + blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<N)
		mapitfoo(idx,out,in);
  }
}
 
// main routine that executes on the host
int main(void)
{
  int *in_h, *in_d, *out_h, *out_d;  // Pointer to host & device arrays
  const int N = 10;  // Number of elements in arrays


  size_t size = N * sizeof(int);
  in_h = (int *)malloc(size);        // Allocate array on host
  out_h = (int *)malloc(size);        // Allocate array on host

  hipMalloc((void **) &in_d, size);   // Allocate array on device
  hipMalloc((void **) &out_d, size);   // Allocate array on device


  // Initialize host array and copy it to CUDA device
  for (int i=0; i<N; i++) in_h[i] = i;
  hipMemcpy(in_d, in_h, size, hipMemcpyHostToDevice);

  // Do calculation on device:
  int block_size = 32; //TODO: get this number from device info
  int n_blocks = 1; //TODO: determine number of SM's
  square_array <<< n_blocks, block_size >>> (N, out_d, in_d);
  // Retrieve result from device and store it in host array

  hipMemcpy(out_h, out_d, sizeof(int)*N, hipMemcpyDeviceToHost);
  
  // Print results
  for (int i=0; i<N; i++) printf("%d %d %d\n", i, in_h[i], out_h[i]);
  // Cleanup
  free(in_h); 
  free(out_h);
  hipFree(in_d);
  hipFree(out_d);
}
